#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
 
#include "cudaUtility.h"



// gpuSegOverlay
template<bool filter_linear, bool mask_only>
__global__ void gpuSegOverlay( float4* input, const int in_width, const int in_height,
						 float4* output, const int out_width, const int out_height,
						 float4* class_colors, uint8_t* scores, const int2 scores_dim )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= out_width || y >= out_height )
		return;

	const float px = float(x) / float(out_width);
	const float py = float(y) / float(out_height);

	#define LOOKUP_CLASS_MAP(ix, iy)	scores[iy * scores_dim.x + ix]

	// point or linear filtering mode
	if( !filter_linear )
	{
		// calculate coordinates in scores cell
		const float cx = px * float(scores_dim.x);	
		const float cy = py * float(scores_dim.y);

		const int x1 = int(cx);
		const int y1 = int(cy);

		// get the class ID of this cell
		const uint8_t classIdx = LOOKUP_CLASS_MAP(x1, y1);

		// find the color of this class
		const float4 classColor = class_colors[classIdx];

		// output the pixel
		if( mask_only )
		{
			// only draw the segmentation mask
			output[y * out_width + x] = classColor;
		}
		else
		{
			// alpha blend with input image
			const int x_in = px * float(in_width);
			const int y_in = py * float(in_height);

			const float4 px_in = input[y_in * in_width + x_in];

			const float alph = classColor.w / 255.0f;
			const float inva = 1.0f - alph;

			output[y * out_width + x] = make_float4(
				alph * classColor.x + inva * px_in.x,
				alph * classColor.y + inva * px_in.y,
				alph * classColor.z + inva * px_in.z,
				255.0f);
		}
	}
	else
	{
		// calculate coordinates in scores cell
		const float bx = (px * float(scores_dim.x)) - 0.5f;
		const float by = (py * float(scores_dim.y)) - 0.5f;

		const float cx = bx < 0.0f ? 0.0f : bx;
		const float cy = by < 0.0f ? 0.0f : by;

		const int x1 = int(cx);
		const int y1 = int(cy);
			
		const int x2 = x1 >= scores_dim.x - 1 ? x1 : x1 + 1;	// bounds check
		const int y2 = y1 >= scores_dim.y - 1 ? y1 : y1 + 1;
		
		const uchar4 classIdx = make_uchar4(LOOKUP_CLASS_MAP(x1, y1),
									 LOOKUP_CLASS_MAP(x2, y1),
									 LOOKUP_CLASS_MAP(x2, y2),
									 LOOKUP_CLASS_MAP(x1, y2));

		const float4 cc[] = { class_colors[classIdx.x],
						  class_colors[classIdx.y],
						  class_colors[classIdx.z],
						  class_colors[classIdx.w] };

		// compute bilinear weights
		const float x1d = cx - float(x1);
		const float y1d = cy - float(y1);

		const float x1f = 1.0f - x1d;
		const float y1f = 1.0f - y1d;

		const float x2f = 1.0f - x1f;
		const float y2f = 1.0f - y1f;

		const float x1y1f = x1f * y1f;
		const float x1y2f = x1f * y2f;
		const float x2y1f = x2f * y1f;
		const float x2y2f = x2f * y2f;

		const float4 classColor = make_float4(
			cc[0].x * x1y1f + cc[1].x * x2y1f + cc[2].x * x2y2f + cc[3].x * x1y2f,
			cc[0].y * x1y1f + cc[1].y * x2y1f + cc[2].y * x2y2f + cc[3].y * x1y2f,
			cc[0].z * x1y1f + cc[1].z * x2y1f + cc[2].z * x2y2f + cc[3].z * x1y2f,
			cc[0].w * x1y1f + cc[1].w * x2y1f + cc[2].w * x2y2f + cc[3].w * x1y2f );

		// output the pixel
		if( mask_only )
		{
			// only draw the segmentation mask
			output[y * out_width + x] = classColor;
		}
		else
		{
			// alpha blend with input image
			const int x_in = px * float(in_width);
			const int y_in = py * float(in_height);

			const float4 px_in = input[y_in * in_width + x_in];

			const float alph = classColor.w / 255.0f;
			const float inva = 1.0f - alph;

			output[y * out_width + x] = make_float4(
				alph * classColor.x + inva * px_in.x,
				alph * classColor.y + inva * px_in.y,
				alph * classColor.z + inva * px_in.z,
				255.0f);
		}

	}
}

// cudaPreImageNet
hipError_t cudaSegOverlay( float4* input, uint32_t in_width, uint32_t in_height,
				        float4* output, uint32_t out_width, uint32_t out_height,
					   float4* class_colors, uint8_t* scores, const int2& scores_dim,
					   bool filter_linear, bool mask_only, hipStream_t stream )
{
	if( !output )
		return hipErrorInvalidDevicePointer;

	if( out_width == 0 || out_height == 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(out_width,blockDim.x), iDivUp(out_height,blockDim.y));

	#define LAUNCH_OVERLAY_KERNEL(filter, mask) gpuSegOverlay<filter, mask><<<gridDim, blockDim, 0, stream>>>(input, in_width, in_height, output, out_width, out_height, class_colors, scores, scores_dim)
	
	if( filter_linear )
	{
		if( mask_only )
			LAUNCH_OVERLAY_KERNEL(true, true);
		else
			LAUNCH_OVERLAY_KERNEL(true, false);
	}
	else
	{
		if( mask_only )
			LAUNCH_OVERLAY_KERNEL(false, true);
		else
			LAUNCH_OVERLAY_KERNEL(false, false);
	}

	return CUDA(hipGetLastError());
}




