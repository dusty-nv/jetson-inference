#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaRGB.h"

//-------------------------------------------------------------------------------------------------------------------------

__global__ void RGBToRGBAf(uchar3* srcImage,
                           float4* dstImage,
                           uint32_t width,
                           uint32_t height)
{
    int x, y, pixel;

    x = (blockIdx.x * blockDim.x) + threadIdx.x;
    y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    pixel = y * width + x;

    if (x >= width)
        return; 

    if (y >= height)
        return;

//	printf("cuda thread %i %i  %i %i pixel %i \n", x, y, width, height, pixel);
        
    const float  s  = 1.0f;
    const uchar3 px = srcImage[pixel];
    
    dstImage[pixel] = make_float4(px.x * s, px.y * s, px.z * s, 255.0f * s);
}

__global__ void RGBAfToRGB(float4 *srcImage,
                           uchar3 *dstImage,
                           uint32_t width,
                           uint32_t height)
{
    int x, y, pixel;

    x = (blockIdx.x * blockDim.x) + threadIdx.x;
    y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    pixel = y * width + x;

    if (x >= width)
        return; 

    if (y >= height)
        return;

    const float4 px = srcImage[pixel];

    dstImage[pixel] = make_uchar3(px.x, px.y, px.z);
}

hipError_t cudaRGBToRGBAf( uchar3* srcDev, float4* destDev, size_t width, size_t height )
{
    if( !srcDev || !destDev )
        return hipErrorInvalidDevicePointer;

    const dim3 blockDim(8,8,1);
    const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y), 1);

    RGBToRGBAf<<<gridDim, blockDim>>>( srcDev, destDev, width, height );

    return CUDA(hipGetLastError());
}

hipError_t cudaRGBAfToRGB( float4* srcDev, uchar3* destDev, size_t width, size_t height )
{
    if( !srcDev || !destDev )
        return hipErrorInvalidDevicePointer;

    const dim3 blockDim(8,8,1);
    const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y), 1);

    RGBAfToRGB<<<gridDim, blockDim>>>( srcDev, destDev, width, height );
    
    return CUDA(hipGetLastError());
}
