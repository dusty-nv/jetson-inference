#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
 
#include "cudaUtility.h"
#include "cudaVector.h"
#include "segNet.h"


// gpuSegOverlay
template<typename T, bool filter_linear, bool mask_only>
__global__ void gpuSegOverlay( T* input, const int in_width, const int in_height,
						 T* output, const int out_width, const int out_height,
						 float4* class_colors, uint8_t* scores, const int2 scores_dim )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= out_width || y >= out_height )
		return;

	const float px = float(x) / float(out_width);
	const float py = float(y) / float(out_height);

	#define LOOKUP_CLASS_MAP(ix, iy)	scores[iy * scores_dim.x + ix]

	// point or linear filtering mode
	if( !filter_linear )
	{
		// calculate coordinates in scores cell
		const float cx = px * float(scores_dim.x);	
		const float cy = py * float(scores_dim.y);

		const int x1 = int(cx);
		const int y1 = int(cy);

		// get the class ID of this cell
		const uint8_t classIdx = LOOKUP_CLASS_MAP(x1, y1);

		// find the color of this class
		const float4 classColor = class_colors[classIdx];

		// output the pixel
		if( mask_only )
		{
			// only draw the segmentation mask
			output[y * out_width + x] = make_vec<T>(classColor.x, classColor.y, classColor.z, 255);
		}
		else
		{
			// alpha blend with input image
			const int x_in = px * float(in_width);
			const int y_in = py * float(in_height);

			const T px_in = input[y_in * in_width + x_in];

			const float alph = classColor.w / 255.0f;
			const float inva = 1.0f - alph;

			output[y * out_width + x] = make_vec<T>(
				alph * classColor.x + inva * px_in.x,
				alph * classColor.y + inva * px_in.y,
				alph * classColor.z + inva * px_in.z,
				255.0f);
		}
	}
	else
	{
		// calculate coordinates in scores cell
		const float bx = (px * float(scores_dim.x)) - 0.5f;
		const float by = (py * float(scores_dim.y)) - 0.5f;

		const float cx = bx < 0.0f ? 0.0f : bx;
		const float cy = by < 0.0f ? 0.0f : by;

		const int x1 = int(cx);
		const int y1 = int(cy);
			
		const int x2 = x1 >= scores_dim.x - 1 ? x1 : x1 + 1;	// bounds check
		const int y2 = y1 >= scores_dim.y - 1 ? y1 : y1 + 1;
		
		const uchar4 classIdx = make_uchar4(LOOKUP_CLASS_MAP(x1, y1),
									 LOOKUP_CLASS_MAP(x2, y1),
									 LOOKUP_CLASS_MAP(x2, y2),
									 LOOKUP_CLASS_MAP(x1, y2));

		const float4 cc[] = { class_colors[classIdx.x],
						  class_colors[classIdx.y],
						  class_colors[classIdx.z],
						  class_colors[classIdx.w] };

		// compute bilinear weights
		const float x1d = cx - float(x1);
		const float y1d = cy - float(y1);

		const float x1f = 1.0f - x1d;
		const float y1f = 1.0f - y1d;

		const float x2f = 1.0f - x1f;
		const float y2f = 1.0f - y1f;

		const float x1y1f = x1f * y1f;
		const float x1y2f = x1f * y2f;
		const float x2y1f = x2f * y1f;
		const float x2y2f = x2f * y2f;

		const float4 classColor = make_float4(
			cc[0].x * x1y1f + cc[1].x * x2y1f + cc[2].x * x2y2f + cc[3].x * x1y2f,
			cc[0].y * x1y1f + cc[1].y * x2y1f + cc[2].y * x2y2f + cc[3].y * x1y2f,
			cc[0].z * x1y1f + cc[1].z * x2y1f + cc[2].z * x2y2f + cc[3].z * x1y2f,
			cc[0].w * x1y1f + cc[1].w * x2y1f + cc[2].w * x2y2f + cc[3].w * x1y2f );

		// output the pixel
		if( mask_only )
		{
			// only draw the segmentation mask
			output[y * out_width + x] = make_vec<T>(classColor.x, classColor.y, classColor.z, 255);
		}
		else
		{
			// alpha blend with input image
			const int x_in = px * float(in_width);
			const int y_in = py * float(in_height);

			const T px_in = input[y_in * in_width + x_in];

			const float alph = classColor.w / 255.0f;
			const float inva = 1.0f - alph;

			output[y * out_width + x] = make_vec<T>(
				alph * classColor.x + inva * px_in.x,
				alph * classColor.y + inva * px_in.y,
				alph * classColor.z + inva * px_in.z,
				255.0f);
		}

	}
}

// cudaSegOverlay
hipError_t cudaSegOverlay( void* input, uint32_t in_width, uint32_t in_height,
				        void* output, uint32_t out_width, uint32_t out_height, imageFormat format,
					   float4* class_colors, uint8_t* scores, const int2& scores_dim,
					   bool filter_linear, bool mask_only, hipStream_t stream )
{
	if( !output )
		return hipErrorInvalidDevicePointer;

	if( out_width == 0 || out_height == 0 )
		return hipErrorInvalidValue;

	if( !imageFormatIsRGB(format) )
	{
		LogError(LOG_TRT "segNet -- unsupported image format (%s)\n", imageFormatToStr(format));
		LogError(LOG_TRT "          supported formats are:\n");
		LogError(LOG_TRT "              * rgb8\n");		
		LogError(LOG_TRT "              * rgba8\n");		
		LogError(LOG_TRT "              * rgb32f\n");		
		LogError(LOG_TRT "              * rgba32f\n");

		return hipErrorInvalidValue;
	}

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(out_width,blockDim.x), iDivUp(out_height,blockDim.y));

	#define LAUNCH_OVERLAY_KERNEL(type, filter, mask) gpuSegOverlay<type, filter, mask><<<gridDim, blockDim, 0, stream>>>((type*)input, in_width, in_height, (type*)output, out_width, out_height, class_colors, scores, scores_dim)
	
	#define LAUNCH_OVERLAY(filter, mask) 				\
		if( format == IMAGE_RGB8 ) {					\
			LAUNCH_OVERLAY_KERNEL(uchar3, filter, mask);	\
		}										\
		else if( format == IMAGE_RGBA8 ) {				\
			LAUNCH_OVERLAY_KERNEL(uchar4, filter, mask);	\
		}										\
		else if( format == IMAGE_RGB32F ) {			\
			LAUNCH_OVERLAY_KERNEL(float3, filter, mask);	\
		}										\
		else if( format == IMAGE_RGBA32F )	{			\
			LAUNCH_OVERLAY_KERNEL(float4, filter, mask); \
		}										

	if( filter_linear )
	{
		if( mask_only )
		{
			LAUNCH_OVERLAY(true, true)
		}
		else
		{
			LAUNCH_OVERLAY(true, false)
		}
	}
	else
	{
		if( mask_only )
		{
			LAUNCH_OVERLAY(false, true)
		}
		else
		{
			LAUNCH_OVERLAY(false, false)
		}
	}

	return CUDA(hipGetLastError());
}




