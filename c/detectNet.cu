#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "detectNet.h"
#include "cudaUtility.h"



template<typename T>
__global__ void gpuDetectionOverlay( T* input, T* output, int width, int height, detectNet::Detection* detections, int numDetections, float4* colors ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= width || y >= height )
		return;

	const T px_in = input[ y * width + x ];
	T px_out = px_in;
	
	const float fx = x;
	const float fy = y;
	
	for( int n=0; n < numDetections; n++ )
	{
		const detectNet::Detection det = detections[n];

		// check if this pixel is inside the bounding box
		if( fx >= det.Left && fx <= det.Right && fy >= det.Top && fy <= det.Bottom )
		{
			const float4 color = colors[det.ClassID];	

			const float alpha = color.w / 255.0f;
			const float ialph = 1.0f - alpha;

			px_out.x = alpha * color.x + ialph * px_out.x;
			px_out.y = alpha * color.y + ialph * px_out.y;
			px_out.z = alpha * color.z + ialph * px_out.z;
		}
	}
	
	output[y * width + x] = px_out;	 
}


template<typename T>
__global__ void gpuDetectionOverlayBox( T* input, T* output, int imgWidth, int imgHeight, int x0, int y0, int boxWidth, int boxHeight, const float4 color ) 
{
	const int box_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int box_y = blockIdx.y * blockDim.y + threadIdx.y;

	if( box_x >= boxWidth || box_y >= boxHeight )
		return;

	const int x = box_x + x0;
	const int y = box_y + y0;

	if( x >= imgWidth || y >= imgHeight )
		return;

	T px = input[ y * imgWidth + x ];

	const float alpha = color.w / 255.0f;
	const float ialph = 1.0f - alpha;

	px.x = alpha * color.x + ialph * px.x;
	px.y = alpha * color.y + ialph * px.y;
	px.z = alpha * color.z + ialph * px.z;
	
	output[y * imgWidth + x] = px;
}

template<typename T>
hipError_t launchDetectionOverlay( T* input, T* output, uint32_t width, uint32_t height, detectNet::Detection* detections, int numDetections, float4* colors )
{
	if( !input || !output || width == 0 || height == 0 || !detections || numDetections == 0 || !colors )
		return hipErrorInvalidValue;
			
	// if input and output are the same image, then we can use the faster method
	// which draws 1 box per kernel, but doesn't copy pixels that aren't inside boxes
	if( input == output )
	{
		for( int n=0; n < numDetections; n++ )
		{
			const int boxWidth = (int)detections[n].Width();
			const int boxHeight = (int)detections[n].Height();

			// launch kernel
			const dim3 blockDim(8, 8);
			const dim3 gridDim(iDivUp(boxWidth,blockDim.x), iDivUp(boxHeight,blockDim.y));

			gpuDetectionOverlayBox<T><<<gridDim, blockDim>>>(input, output, width, height, (int)detections[n].Left, (int)detections[n].Top, boxWidth, boxHeight, colors[detections[n].ClassID]); 
		}
	}
	else
	{
		// launch kernel
		const dim3 blockDim(8, 8);
		const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

		gpuDetectionOverlay<T><<<gridDim, blockDim>>>(input, output, width, height, detections, numDetections, colors); 
	}

	return hipGetLastError();
}

hipError_t cudaDetectionOverlay( void* input, void* output, uint32_t width, uint32_t height, imageFormat format, detectNet::Detection* detections, int numDetections, float4* colors )
{
	if( format == IMAGE_RGB8 )
		return launchDetectionOverlay<uchar3>((uchar3*)input, (uchar3*)output, width, height, detections, numDetections, colors); 
	else if( format == IMAGE_RGBA8 )
		return launchDetectionOverlay<uchar4>((uchar4*)input, (uchar4*)output, width, height, detections, numDetections, colors);  
	else if( format == IMAGE_RGB32F )
		return launchDetectionOverlay<float3>((float3*)input, (float3*)output, width, height, detections, numDetections, colors);  
	else if( format == IMAGE_RGBA32F )
		return launchDetectionOverlay<float4>((float4*)input, (float4*)output, width, height, detections, numDetections, colors); 
	else
		return hipErrorInvalidValue;
}

