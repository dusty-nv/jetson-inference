#include "hip/hip_runtime.h"
/*
 * http://github.com/dusty-nv/jetson-inference
 */
 
#include "cudaUtility.h"



// gpuPreImageNet
__global__ void gpuPreImageNet( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = oWidth * oHeight;
	
	if( x >= oWidth || y >= oHeight )
		return;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.z, px.y, px.x);
	
	output[n * 0 + y * oWidth + x] = bgr.x;
	output[n * 1 + y * oWidth + x] = bgr.y;
	output[n * 2 + y * oWidth + x] = bgr.z;
}


// cudaPreImageNet
hipError_t cudaPreImageNet( float4* input, size_t inputWidth, size_t inputHeight,
				         float* output, size_t outputWidth, size_t outputHeight )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNet<<<gridDim, blockDim>>>(scale, input, inputWidth, output, outputWidth, outputHeight);

	return CUDA(hipGetLastError());
}




// gpuPreImageNetMean
__global__ void gpuPreImageNetMean( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight, float3 mean_value )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = oWidth * oHeight;
	
	if( x >= oWidth || y >= oHeight )
		return;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.z - mean_value.x, px.y - mean_value.y, px.x - mean_value.z);
	
	output[n * 0 + y * oWidth + x] = bgr.x;
	output[n * 1 + y * oWidth + x] = bgr.y;
	output[n * 2 + y * oWidth + x] = bgr.z;
}


// cudaPreImageNetMean
hipError_t cudaPreImageNetMean( float4* input, size_t inputWidth, size_t inputHeight,
				             float* output, size_t outputWidth, size_t outputHeight, const float3& mean_value )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetMean<<<gridDim, blockDim>>>(scale, input, inputWidth, output, outputWidth, outputHeight, mean_value);

	return CUDA(hipGetLastError());
}

// gpuPreImageNetMean
__global__ void gpuPreImageNetMean( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight, float* mean_binary)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = oWidth * oHeight;
	
	if( x >= oWidth || y >= oHeight )
		return;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	//const float3 mx  = mean_binary[ dy * iWidth + dx ];
	//const float3 bgr = make_float3(px.z - mx.x, px.y - mx.y, px.x - mx.z);
	
	output[n * 0 + y * oWidth + x] = px.z - mean_binary[n * 0 + y * oWidth + x];
	output[n * 1 + y * oWidth + x] = px.y - mean_binary[n * 1 + y * oWidth + x];
	output[n * 2 + y * oWidth + x] = px.x - mean_binary[n * 2 + y * oWidth + x];
}


// cudaPreImageNetMean
hipError_t cudaPreImageNetMean( float4* input, size_t inputWidth, size_t inputHeight,
				             float* output, size_t outputWidth, size_t outputHeight, float* mean_binary )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetMean<<<gridDim, blockDim>>>(scale, input, inputWidth, output, outputWidth, outputHeight, mean_binary);

	return CUDA(hipGetLastError());
}


