#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
 
#include "cudaUtility.h"


// rgbaToGray
__device__ inline float rgbaToGray( const float4& rgba )
{
	return rgba.x * 0.2989f + rgba.y * 0.5870f + rgba.z * 0.1140f;
}


// normalize to [-1,1]
__device__ inline float norm1( float value )
{
	return value / 255.0f * 2.0f - 1.0f;
}


// gpuPreHomographyNet
__global__ void gpuPreHomographyNet( float2 scale, float4* in_A, float4* in_B, int in_width, 
							  float* output, int out_width, int out_height )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = out_width * out_height;
	
	if( x >= out_width || y >= out_height )
		return;

	// scale coordinates to input
	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	// convert inputs to grayscale
	const int in_idx = dy * in_width + dx;

	const float4 rgba_A = in_A[in_idx];
	const float4 rgba_B = in_B[in_idx];

	const float gray_A = rgbaToGray(rgba_A);
	const float gray_B = rgbaToGray(rgba_B);

	const float norm_A = norm1(gray_A);
	const float norm_B = norm1(gray_B);

	// concatenate the images
	output[n * 0 + y * out_width + x] = norm_A;
	output[n * 1 + y * out_width + x] = norm_B;
}


// cudaPreHomographyNet
hipError_t cudaPreHomographyNet( float4* inputA, float4* inputB, size_t inputWidth, size_t inputHeight,
				         	    float* output, size_t outputWidth, size_t outputHeight,
					         hipStream_t stream )
{
	if( !inputA || !inputB || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreHomographyNet<<<gridDim, blockDim, 0, stream>>>(scale, inputA, inputB, inputWidth, output, outputWidth, outputHeight);

	return CUDA(hipGetLastError());
}

