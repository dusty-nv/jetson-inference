#include "hip/hip_runtime.h"
// Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
// Full license terms provided in LICENSE.md file.

#include "internal_utils.h"
#include <hip/hip_fp16.h>

// Check async error.
// Sync and get kernel status in Debug builds.
#ifndef NDEBUG
    #define SYNC_AND_CHECK_STREAM(stream) do {          \
    hipError_t status = hipStreamSynchronize(stream); \
    if (status != hipSuccess)                          \
        return status;                                  \
}while(false)
#else
    #define SYNC_AND_CHECK_STREAM(stream)
#endif

#define CHECKK(stream) do {                  \
    hipError_t status = hipGetLastError(); \
    if (status != hipSuccess)               \
        return status;                       \
    SYNC_AND_CHECK_STREAM(stream);           \
}while(false)

namespace redtail { namespace tensorrt
{

using namespace nvinfer1;

static const int kMaxGridSizeY = 65535;
static const int kMaxGridSizeZ = 65535;

// -----------------------------------------------------------------
// Helper function to get block count.
// -----------------------------------------------------------------
static uint32_t getBlockCount(uint32_t total_size, uint32_t block_size)
{
    uint32_t res = (total_size + block_size - 1) / block_size;
    assert(res > 0);
    assert((size_t)res * block_size >= total_size);
    return res;
}

// REVIEW alexeyk: kernels are not optimized for now.

// -----------------------------------------------------------------
// Cost volume kernels.
// -----------------------------------------------------------------
template<typename T>
__global__ void costVolumeCopyKernel(const T* src, int32_t c, int32_t h, int32_t w, int32_t disp, T* dst)
{
    assert(src != nullptr);
    assert(dst != nullptr);

    const uint32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t iz = blockIdx.z * blockDim.z + threadIdx.z;
    if (ix >= w || iy >= h || iz >= c)
        return;
    const size_t isrc   = iz * h * w + iy * w + ix;
    const size_t stride = 2 * c * h * w;
    T  val  = src[isrc];
    T* pdst = dst + isrc;
    for (int32_t idst = 0; idst < disp; idst++)
    {
        *pdst = val;
        pdst += stride;
    }
}

template<typename T>
__global__ void costVolumeCopyPadKernel(const T* src, int32_t c, int32_t h, int32_t w, int32_t disp, T* dst)
{
    assert(src != nullptr);
    assert(dst != nullptr);

    const uint32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t iz = blockIdx.z * blockDim.z + threadIdx.z;
    if (ix >= w || iy >= h || iz >= c)
        return;
    const size_t isrc    = iz * h * w + iy * w + ix;
    size_t       stride  = c * h * w;
    const size_t idst    = isrc + stride;
    stride *= 2;

    T* pdst = dst + idst;
    for (int32_t pad = 0; pad < disp; pad++)
    {
        if (ix < pad)
            *pdst = 0;
        else
            *pdst = src[isrc - pad];
        pdst += stride;
    }
}

template<typename T>
__global__ void costVolumeKernel(const T* left, const T* right, int32_t c, int32_t h, int32_t w, int32_t disp, T* dst)
{
    assert(left  != nullptr);
    assert(right != nullptr);
    assert(dst   != nullptr);

    const uint32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t iz = blockIdx.z * blockDim.z + threadIdx.z;
    if (ix >= w || iy >= h || iz >= c)
        return;

    // Setup initial indices.
    size_t       stride = c * h * w;
    // Left and right source is the same.
    const size_t ileft  = iz * h * w + iy * w + ix;
    T*           pdst_l = dst + ileft;
    const size_t iright = ileft;
    // Right destination is offset by 1 in c dimension.
    T*           pdst_r = dst + iright + stride;
    // Final stride is 2 in c dimension.
    stride *= 2;

    T  val_l  = left[ileft];
    for (int32_t pad = 0; pad < disp; pad++)
    {
        if (ix < pad)
            *pdst_r = 0;
        else
            *pdst_r = right[iright - pad];
        *pdst_l = val_l;
        pdst_l += stride;
        pdst_r += stride;
    }
}

template<>
hipError_t CudaKernels::computeCostVolume(DataType data_type, const float* left, const float* right, Dims in_dims, 
                                           float* cost_vol, Dims out_dims, hipStream_t stream)
{
    assert(data_type == DataType::kFLOAT);
    assert(in_dims.nbDims  == 3);
    assert(out_dims.nbDims == 4);

    dim3 b_dim{16, 16, 1};
    dim3 g_dim;
    g_dim.x = getBlockCount(in_dims.d[2], b_dim.x);
    g_dim.y = getBlockCount(in_dims.d[1], b_dim.y);
    g_dim.z = getBlockCount(in_dims.d[0], b_dim.z);

    // REVIEW alexeyk: using 2 kernels instead of one as it's not yet optimized so 2 kernels are faster.
    // REVIEW alexeyk: optimize, see gld_efficiency,gst_efficiency,gld_transactions,gst_transactions.
    // costVolumeKernel<<<g_dim, b_dim, 0, stream>>>(left, right, in_dims.d[0], in_dims.d[1], in_dims.d[2], out_dims.d[0],
    //                                               cost_vol);
    costVolumeCopyKernel<<<g_dim, b_dim, 0, stream>>>(left, in_dims.d[0], in_dims.d[1], in_dims.d[2], out_dims.d[0],
                                                      cost_vol);
    CHECKK(stream);
    costVolumeCopyPadKernel<<<g_dim, b_dim, 0, stream>>>(right, in_dims.d[0], in_dims.d[1], in_dims.d[2], out_dims.d[0],
                                                         cost_vol);
    CHECKK(stream);
    return hipSuccess;
}

// -----------------------------------------------------------------
// Correlation cost volume kernels.
// -----------------------------------------------------------------

// FP32, NCHW kernel.
template<typename T>
__global__ void corrCostVolumeKernel(const T* left, const T* right, int32_t c, int32_t h, int32_t w, int32_t disp, T* dst)
{
    assert(left  != nullptr);
    assert(right != nullptr);
    assert(dst   != nullptr);

    const uint32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix >= w || iy >= h)
        return;

    uint32_t pad = blockIdx.z;
    assert(pad < disp);
    size_t stride = h * w;

    T val = 0;
    if (ix >= pad)
    {
        const T* pl = left  + iy * w + ix;
        const T* pr = right + iy * w + ix - pad;
        for (int32_t i = 0; i < c; i++)
        {
            val += *pl * (*pr);
            pl  += stride;
            pr  += stride;
        }
    }

    // Disparity feature maps are arranged from to min to max.
    size_t idst = pad * h * w + iy * w + ix;
    dst[idst] = val;
}

// FP16, NC2HW2 kernel.
__global__ void corrCostVolumeFP16NC2HW2Kernel(const float* left, const float* right, int32_t c, int32_t h, int32_t w, int32_t disp, float* dst)
{
    assert(left  != nullptr);
    assert(right != nullptr);
    assert(dst   != nullptr);

    const uint32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix >= w || iy >= h)
        return;

    uint32_t pad = 2 * blockIdx.z;
    assert(pad < disp);
    size_t stride = h * w;

    // REVIEW alexeyk: using FP32 arithmetic for better precision. FP16 works fine too
    // but does not give any perf increase and causes slight loss in accuracy.
    // __half2 val1{0, 0};
    // __half2 val2{0, 0};
    float val1 = 0;
    float val2 = 0;
    if (ix >= pad)
    {
        const float* pl = left  + iy * w + ix;
        const float* pr = right + iy * w + ix - pad;
        for (int32_t i = 0; i < (c + 1) / 2; i++)
        {
            // auto l  = *(__half2*)pl;
            // auto r1 = *(__half2*)pr;
            // auto r2 = ix >= pad + 1 ? *(__half2*)(pr - 1) : __half2{0, 0};
            // val1 = __hfma2(l, r1, val1);
            // val2 = __hfma2(l, r2, val2);
            auto l  = __half22float2(*(__half2*)pl);
            auto r1 = __half22float2(*(__half2*)pr);
            auto r2 = ix >= pad + 1 ? __half22float2(*(__half2*)(pr - 1)) : float2{0, 0};
            val1 += l.x * r1.x + l.y * r1.y;
            val2 += l.x * r2.x + l.y * r2.y;
            pl  += stride;
            pr  += stride;
        }
    }

    // Disparity feature maps are arranged from to min to max.
    size_t idst = blockIdx.z * h * w + iy * w + ix;
    // auto val  = __half2(__hadd(val1.x, val1.y), __hadd(val2.x, val2.y));
    auto val  = __half2((__half)val1, (__half)val2);
    dst[idst] = *(float*)&val;
}

template<>
hipError_t CudaKernels::computeCorrCostVolume(DataType data_type, const float* left, const float* right, Dims in_dims, 
                                               float* cost_vol, Dims out_dims, hipStream_t stream)
{
    assert(data_type == DataType::kFLOAT || data_type == DataType::kHALF);
    assert(in_dims.nbDims  == 3);
    assert(out_dims.nbDims == 3);

    if (data_type == DataType::kFLOAT)
    {
        dim3 b_dim{16, 16, 1};
        dim3 g_dim;
        g_dim.x = getBlockCount(in_dims.d[2],  b_dim.x);
        g_dim.y = getBlockCount(in_dims.d[1],  b_dim.y);
        // Each block handles a particular disparity.
        g_dim.z = out_dims.d[0];

        corrCostVolumeKernel<<<g_dim, b_dim, 0, stream>>>(left, right, in_dims.d[0], in_dims.d[1], in_dims.d[2], out_dims.d[0],
                                                          cost_vol);
        CHECKK(stream);
    }
    else if (data_type == DataType::kHALF)
    {
        dim3 b_dim{16, 16, 1};
        dim3 g_dim;
        g_dim.x = getBlockCount(in_dims.d[2], b_dim.x);
        g_dim.y = getBlockCount(in_dims.d[1], b_dim.y);
        // Each block handles 2 disparity values.
        g_dim.z = (out_dims.d[0] + 1) / 2;

        corrCostVolumeFP16NC2HW2Kernel<<<g_dim, b_dim, 0, stream>>>(left, right, in_dims.d[0], in_dims.d[1], in_dims.d[2], out_dims.d[0],
                                                                    cost_vol);
        CHECKK(stream);
    }
    return hipSuccess;
}

// -----------------------------------------------------------------
// Some convolution-related kernels.
// -----------------------------------------------------------------
template<typename T>
__global__ void addDBiasTo3DConvKernel(const T* bias, int32_t c, int32_t d, int32_t h, int32_t w, T* conv)
{
    assert(bias != nullptr);
    assert(conv != nullptr);

    const uint32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t iz = blockIdx.z * blockDim.z + threadIdx.z;
    if (ix >= w || iy >= h || iz >= d * c)
        return;

    int32_t cur_d     = iz % d;
    const size_t idst = iz * h * w + iy * w + ix;

    conv[idst] += bias[cur_d];
}

template<>
hipError_t CudaKernels::addDBiasTo3DConv(const float* bias, Dims bias_dims, float* conv, Dims conv_dims, hipStream_t stream)
{
    assert(bias_dims.nbDims == 5);
    assert(conv_dims.nbDims == 4);
    // REVIEW alexeyk: minibatch size 1 for now.
    assert(bias_dims.d[0] == 1);
    assert(bias_dims.d[2] == conv_dims.d[1]);
    UNUSEDR(bias_dims);

    dim3 b_dim{16, 16, 1};
    dim3 g_dim;
    g_dim.x = getBlockCount(conv_dims.d[3], b_dim.x);
    g_dim.y = getBlockCount(conv_dims.d[2], b_dim.y);
    g_dim.z = getBlockCount(conv_dims.d[0] * conv_dims.d[1], b_dim.z);
    // REVIEW alexeyk: no block striding for now.
    assert(g_dim.y <= kMaxGridSizeY);
    assert(g_dim.z <= kMaxGridSizeZ);
    UNUSEDR(kMaxGridSizeY);
    UNUSEDR(kMaxGridSizeZ);

    addDBiasTo3DConvKernel<<<g_dim, b_dim, 0, stream>>>(bias, conv_dims.d[0], conv_dims.d[1], conv_dims.d[2], conv_dims.d[3], conv);
    CHECKK(stream);

    return hipSuccess;
}

// -----------------------------------------------------------------
// Conversion kernels.
// -----------------------------------------------------------------
__global__ void fp32Tofp16Kernel(const float* src, uint16_t* dst, size_t size)
{
    assert(src != nullptr);
    assert(dst != nullptr);

    const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size)
        return;

    __half val(src[tid]);
    dst[tid] = *(uint16_t*)&val;
}

hipError_t CudaKernels::fp32Tofp16(const float* src, uint16_t* dst, size_t size, hipStream_t stream)
{
    dim3 b_dim{256, 1, 1};
    dim3 g_dim;
    g_dim.x = getBlockCount(size, b_dim.x);

    fp32Tofp16Kernel<<<g_dim, b_dim, 0, stream>>>(src, dst, size);
    CHECKK(stream);

    return hipSuccess;
}

__global__ void fp16Tofp32Kernel(const uint16_t* src, float* dst, size_t size)
{
    assert(src != nullptr);
    assert(dst != nullptr);

    const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= size)
        return;

    dst[tid] = (float)(*(__half*)(src + tid));
}

hipError_t CudaKernels::fp16Tofp32(const uint16_t* src, float* dst, size_t size, hipStream_t stream)
{
    dim3 b_dim{256, 1, 1};
    dim3 g_dim;
    g_dim.x = getBlockCount(size, b_dim.x);

    fp16Tofp32Kernel<<<g_dim, b_dim, 0, stream>>>(src, dst, size);
    CHECKK(stream);

    return hipSuccess;
}

} }