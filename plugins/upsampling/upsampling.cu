#include "hip/hip_runtime.h"
#include "upsampling.h"
#include <iostream>
#include <cassert>

// gpu operation for nearest neighbor upsampling
template <typename T>
__global__ void gpuResizeNearestNeighbor( T* input, int nChannels, int iHeight, int iWidth, int oHeight, int oWidth, T* output)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    if( x >= nChannels || y >= oHeight || z >= oWidth )
        return;

    const int dy = y / 2;
    const int dz = z / 2;

    const T px = input[x * iWidth * iHeight + dy * iWidth + dz];

    output[x * oWidth * oHeight + y * oWidth + z] = px;
}


// nearest neighbor upsampling
template <typename T>
hipError_t cudaResizeNearestNeighbor( T* input, int nChannels, int inputHeight, int inputWidth,
                        T* output, hipStream_t stream )
{
    if( !input || !output )
    {
        std::cout << "No input or no output" << std::endl;
        return hipErrorInvalidDevicePointer;
    }

    if( inputWidth == 0 || inputHeight == 0 )
    {
        std::cout << "Width or height is 0" << std::endl;
        return hipErrorInvalidValue;
    }

    // launch kernel
    const dim3 blockDim(1, 16, 16);
    const size_t outputWidth = 2 * inputWidth;
    const size_t outputHeight = 2 * inputHeight;
    const dim3 gridDim(iDivUp(nChannels, blockDim.x), iDivUp(outputHeight, blockDim.y), iDivUp(outputWidth, blockDim.z));

    gpuResizeNearestNeighbor<T><<<gridDim, blockDim, 0, stream>>>(input, nChannels, inputHeight, inputWidth, outputHeight, outputWidth, output);

    return CUDA(hipGetLastError());
}

//gpu operation for bilinear upsampling
template <typename T>
// TODO
__global__ void gpuResizeBilinear( float2 scale, T* input, int iWidth, T* output, int oWidth, int oHeight ) { }

// bilinear upsampling
hipError_t cudaResizeBilinear( float* input, size_t inputWidth, size_t inputHeight,
                        float* output, size_t outputWidth, size_t outputHeight )
{
    // TODO
    return CUDA(hipGetLastError());
}

template hipError_t cudaResizeNearestNeighbor<float>(float*, int, int, int, float*, hipStream_t);
template hipError_t cudaResizeNearestNeighbor<__half>(__half*, int, int, int, __half*, hipStream_t);
