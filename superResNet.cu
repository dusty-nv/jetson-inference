#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
 
#include "cudaUtility.h"



// clip float to [min,max]
static inline __device__ float clip( const float x, float min, float max )
{
	return x > max ? max : x < min ? min : x;
}


// clip vector to [min,max]
static inline __device__ float4 clip( const float4& px, float min, float max )
{
	return make_float4(clip(px.x, min, max),
				    clip(px.y, min, max),
				    clip(px.z, min, max),
				    clip(px.w, min, max));
}


// gpuPreSuperResNet
template<typename T>
__global__ void gpuPreSuperResNet( T* input, int iWidth, float* output, int oWidth, int oHeight, float2 res_scale, float pixel_scale )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = oWidth * oHeight;
	
	if( x >= oWidth || y >= oHeight )
		return;

	const int dx = ((float)x * res_scale.x);
	const int dy = ((float)y * res_scale.y);

	const T px = input[ dy * iWidth + dx ];
	const float3 rgb = make_float3(px.x * pixel_scale, px.y * pixel_scale, px.z * pixel_scale);
	
	output[n * 0 + y * oWidth + x] = rgb.x;
	output[n * 1 + y * oWidth + x] = rgb.y;
	output[n * 2 + y * oWidth + x] = rgb.z;
}


// cudaPreSuperResNet
hipError_t cudaPreSuperResNet( float4* input, size_t inputWidth, size_t inputHeight,
				            float* output, size_t outputWidth, size_t outputHeight,
					       float maxPixelValue, hipStream_t stream )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 res_scale = make_float2( float(inputWidth) / float(outputWidth),
							        float(inputHeight) / float(outputHeight) );

	const float pixel_scale = 1.0f / maxPixelValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreSuperResNet<float4><<<gridDim, blockDim, 0, stream>>>(input, inputWidth, output, outputWidth, outputHeight, res_scale, pixel_scale);

	return CUDA(hipGetLastError());
}


// gpuPostSuperResNet
template<typename T>
__global__ void gpuPostSuperResNet( float* input, int iWidth, int iHeight, T* output, int oWidth, int oHeight, float2 res_scale, float pixel_scale )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = iWidth * iHeight;
	
	if( x >= oWidth || y >= oHeight )
		return;

	const int dx = ((float)x * res_scale.x);
	const int dy = ((float)y * res_scale.y);

	const float4 rgb = clip(make_float4(input[n * 0 + dy * iWidth + dx] * pixel_scale,
							      input[n * 1 + dy * iWidth + dx] * pixel_scale,
							      input[n * 2 + dy * iWidth + dx] * pixel_scale,
							      pixel_scale), 0.0f, pixel_scale);

	output[y * oWidth + x] = rgb;
}


// cudaPostSuperResNet
hipError_t cudaPostSuperResNet( float* input, size_t inputWidth, size_t inputHeight,
				             float4* output, size_t outputWidth, size_t outputHeight,
					        float maxPixelValue, hipStream_t stream )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 res_scale = make_float2( float(inputWidth) / float(outputWidth),
							        float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPostSuperResNet<float4><<<gridDim, blockDim, 0, stream>>>(input, inputWidth, inputHeight, output, outputWidth, outputHeight, res_scale, maxPixelValue);

	return CUDA(hipGetLastError());
}

